#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"
#include <iostream>

__global__ void BFS(unsigned short* old_boards, unsigned short* new_boards, int* board_index, int boards_count, __int16* old_validators, 
	__int16* new_validators, unsigned short* empty_spaces, unsigned short* empty_cells_count, bool is_last)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	while (index < boards_count)
	{
		// get board_start index
		int board_start = index * N * N;
		int empty_index = board_start;

		// find first empty index in board
		while (empty_index < board_start + N * N)
		{
			if (old_boards[empty_index] == 0)
			{
				// get column, row and subboard of current cell
				int row = (empty_index - board_start) / N;
				int column = (empty_index - board_start) % N;
				int subboard = (row / 3) * 3 + (column / 3);

				// create boards with correct values
				for (int value = 1; value <= N; ++value)
				{
					bool flag = true;

					// check row
					int bit = (1 << value) & (old_validators[index * validator_size + row]);
					if (bit != 0)
					{
						flag = false;
					}

					// check column
					bit = (1 << value) & (old_validators[index * validator_size + N + column]);
					if (bit != 0)
					{
						flag = false;
					}

					// check subboard
					bit = (1 << value) & (old_validators[index * validator_size + 2 * N + subboard]);
					if (bit != 0)
					{
						flag = false;
					}

					// if correct then add board to new_boards
					if (flag)
					{
						// get current_board and update shared board_index
						int current_board = atomicAdd(board_index, 1);
						int e_id = 0;
						for (int j = 0; j < N * N; ++j)
						{
							// update new_boards
							new_boards[current_board * N * N + j] = old_boards[board_start + j];
							if (j < validator_size)
							{
								// update validators
								new_validators[current_board * validator_size + j] = old_validators[index * validator_size + j];
							}
							if (is_last && new_boards[current_board * N * N + j] == 0 && (j / N != row || j % N != column))
							{
								// update empty spaces used in DFS
								empty_spaces[e_id] = j;
								e_id++;
							}
						}

						// assign empty cells count
						*empty_cells_count = e_id;

						// assign value to empty cell
						new_boards[current_board * N * N + empty_index - board_start] = value;

						// update validators with added value
						new_validators[current_board * validator_size + row] |= (1 << value);
						new_validators[current_board * validator_size + N + column] |= (1 << value);
						new_validators[current_board * validator_size + 2 * N + subboard] |= (1 << value);
					}
				}

				// empty cell found
				break;
			}
			else
			{
				empty_index++;
			}
		}

		index += gridDim.x * blockDim.x;
	}
}

__global__ void DFS(unsigned short* boards, __int16* validators, int boards_count, unsigned short* empty_spaces, unsigned short* empty_spaces_count, int* sol_found, unsigned short* sol)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	while ((*sol_found) == 0 && index < boards_count)
	{
		int empty_index = 0;

		unsigned short* current_board = boards + index * N * N;
		__int16* currentValidators = validators + index * validator_size;

		while (empty_index >= 0 && empty_index < *empty_spaces_count)
		{
			int cell_id = empty_spaces[empty_index];

			int row = cell_id / N;
			int column = cell_id % N;
			int subboard = (row / 3) * 3 + (column / 3);

			bool flag = false;
			for (int value = current_board[cell_id] + 1; value <= N; ++value)
			{
				int row_flag = (1 << value) & (currentValidators[row]);
				int column_flag = (1 << value) & (currentValidators[N + column]);;
				int subboard_flag = (1 << value) & (currentValidators[2 * N + subboard]);

				if (row_flag == 0 && column_flag == 0 && subboard_flag == 0)
				{
					flag = true;

					current_board[cell_id] = value;
					currentValidators[row] |= (1 << value);
					currentValidators[N + column] |= (1 << value);
					currentValidators[2 * N + subboard] |= (1 << value);

					empty_index++;
					break;

				}
			}

			if (!flag)
			{
				current_board[cell_id] = 0;
				empty_index--;

				if (empty_index >= 0)
				{
					cell_id = empty_spaces[empty_index];

					unsigned short value = current_board[cell_id];
					row = cell_id / N;
					column = cell_id % N;
					subboard = (row / 3) * 3 + (column / 3);

					currentValidators[row] &= ~(1 << value);
					currentValidators[N + column] &= ~(1 << value);
					currentValidators[2 * N + subboard] &= ~(1 << value);
				}
			}
		}

		if (empty_index == *empty_spaces_count)
		{
			*sol_found = 1;

			for (int i = 0; i < N * N; i++) 
			{
				sol[i] = current_board[i];
			}
		}

		index += gridDim.x * blockDim.x;
	}
}

void kernel_BFS(unsigned short* old_boards, unsigned short* new_boards, int* board_index, int boards_count, __int16* old_validators, __int16* new_validators, unsigned short* empty_spaces, unsigned short* empty_cells_count, bool is_last)
{
	BFS <<< blocks_count, threads_count >>> (old_boards, new_boards, board_index, 
		boards_count, old_validators, new_validators, empty_spaces, empty_cells_count, is_last);
	hipDeviceSynchronize();
}

void kernel_DFS(unsigned short* boards, __int16* validators, int boards_count, unsigned short* empty_spaces, unsigned short* empty_spaces_count, int* sol_found, unsigned short* sol)
{
	DFS << < blocks_count, threads_count >> > (boards, validators, boards_count, empty_spaces, empty_spaces_count, sol_found, sol);
	hipDeviceSynchronize();
}